
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <vector>
#include <string>
#include <fstream>
#include <cmath>
#include <ctime>
#include <stack>
#include <sstream>
#include <cstdlib>
#include <iostream>
#include <limits>
#include <algorithm>
#include <climits>
#include <bitset>
#include <set>
#include <sys/time.h>
#include <bits/stdc++.h>
using namespace std;

#define print_to_err(x) do { std::cerr << #x << ": " << x << std::endl; } while (0)


#define bitcount(x) __builtin_popcount(x)

string int2str(int num)
{
  ostringstream ss;
  ss << num;
  return(ss.str());
}

int str2int(string str)
{
  int value;
  istringstream (str)>>value;
  return value;
}


string input_file="";
string output_file="";
vector<long long int> matrix_A_row;
vector<long long int> matrix_A_col;
vector<long long int> matrix_A_data;

vector<long long int> matrix_B;
vector<long long int> matrix_C;
vector<long long int> ptr_vec;


long long int dimension;




void readfile()
{
    ifstream infile;
    char input_file_array[input_file.length()+1];
    for(int i=0;i<input_file.length();i++)
    {
        input_file_array[i]=input_file.at(i);
    }
    input_file_array[input_file.length()]='\0';


    infile.open(input_file_array);
    //Name Line
    string name_line;
    getline(infile,name_line);

    //Dimension Line
    string dimension_line;
    getline(infile,dimension_line);
    // print_to_err(dimension_line);
    int dimension_line_length=int(dimension_line.length());
    int start_dimension_line;
    for(int i=10;i<dimension_line_length;i++)
    {
        if(dimension_line.at(i)!=' ')
        {
            start_dimension_line=i;
            break;
        }
    }
    string dimension_string=dimension_line.substr(start_dimension_line,(dimension_line_length- start_dimension_line));
    // print_to_err(dimension_string);
    istringstream(dimension_string)>>dimension;

    ptr_vec.resize(dimension+1);
    //A
    string garbage_line;
    getline(infile,garbage_line);
    

    long long int temp_inp_a;
    string b_detect; 
    long long int curr_row=(-1);
    long long int count_ptr=0;
    while(true)
    {
        infile>>b_detect;
        if(b_detect=="B")
        {
            for(long long int j=(curr_row+1);j<=dimension;j++)
            {
                //check this
                ptr_vec[j]=count_ptr;
            }
            break;
        }
        istringstream (b_detect)>>temp_inp_a;
        matrix_A_row.push_back(temp_inp_a);

        if(curr_row<temp_inp_a)
        {
            for(long long int j=(curr_row+1);j<=temp_inp_a;j++)
            {
                ptr_vec[j]=count_ptr;
            }
            curr_row=temp_inp_a;
        }

        infile>>temp_inp_a;
        matrix_A_col.push_back(temp_inp_a);
        infile>>temp_inp_a;
        matrix_A_data.push_back(temp_inp_a);
        count_ptr+=1;
    }
    matrix_B.resize(dimension);
    for(int i=0;i<dimension;i++)
    {
        infile>>matrix_B[i];    
    }
    matrix_C.clear();
    matrix_C.resize(dimension,0);
    infile.close();
}

void outfile()
{
    ofstream outfile;
    char output_file_array[output_file.length()+1];
    for(int i=0;i<output_file.length();i++)
    {
        output_file_array[i]=output_file.at(i);
    }
    output_file_array[output_file.length()]='\0';
    outfile.open (output_file_array);
    // else
    // {
    //     outfile.open(output_file_array, std::ios_base::app);
    // }
    // outfile<<"my rank:"<<my_rank<<" I will print "<<int(my_numbers.size())<<" numbers"<<endl;


    // for(int i=0;i<matrix_A_data.size();i++)
    // {
    //     outfile<<matrix_A_row[i]<<" "<<matrix_A_col[i]<<" "<<matrix_A_data[i]<<endl;
    // }

    for (int i = 0; i < dimension; i++)
    {
        outfile<<matrix_C[i]<<endl;
    }
    // outfile<<endl;
    outfile.close();
}


__global__ void spmv_csr_vector_kernel ( long long int num_rows ,long long int * ptr ,long long int * indices ,long long int * data ,long long int * x ,long long int * y)
{

    __shared__ long long int vals [32];
    long long int thread_id = blockDim.x * blockIdx.x + threadIdx.x ; // global thread index
    long long int warp_id = thread_id / 32; // global warp index
    int lane = thread_id & (32 - 1); // thread index within the warp
    // one warp per row
    long long int row = warp_id ;
    //int num_rows = 5;
    if ( row < num_rows ){
        long long int row_start = ptr [ row ];
        long long int row_end = ptr [ row +1];
        // compute running sum per thread
        vals [ threadIdx.x ] = 0;
        for ( long long int jj = row_start + lane ; jj < row_end ; jj += 32)
            vals [ threadIdx.x ] += data [ jj ] * x [ indices [ jj ]];
        // parallel reduction in shared memory
        if ( lane < 16) vals [ threadIdx.x ] += vals [ threadIdx.x + 16];
        if ( lane < 8) vals [ threadIdx.x ] += vals [ threadIdx.x + 8];
        if ( lane < 4) vals [ threadIdx.x ] += vals [ threadIdx.x + 4];
        if ( lane < 2) vals [ threadIdx.x ] += vals [ threadIdx.x + 2];
        if ( lane < 1) vals [ threadIdx.x ] += vals [ threadIdx.x + 1];
        // first thread writes the result
        if ( lane == 0)
            y[ row ] += vals [ threadIdx.x ];
    }

    // if(thread_id==0)
    // {
    //     cout<<"GPU PRINT"<<endl;
    //     for(int i=0; i<11;i++)
    //     {
    //         cout<<indices[i]<<" "<<data[i]<<endl;
    //     }
    //     cout<<endl;
    //     cout<<"num_rows:"<<num_rows<<endl;
    //     cout<<"B is "<<endl;
    //     for(int i=0; i<5;i++)
    //     {
    //         cout<<x[i]<<endl;
    //     }
    //     cout<<"ptr_vec is "<<endl;
    //     for(int i=0; i<6;i++)
    //     {
    //         cout<<ptr[i]<<endl;
    //     }
    // }



}



int main(int argc, char *argv[])
{
    string temp_inpfile(argv[1]);
    input_file=temp_inpfile;
    string temp_outfile(argv[2]);
    output_file=temp_outfile;


    readfile();



    long long int matrix_A_nonzero_elem=(long long int)(matrix_A_data.size());

//    for(int i=0; i<matrix_A_nonzero_elem;i++)
//    {
//        cout<<matrix_A_row[i]<<" "<<matrix_A_col[i]<<" "<<matrix_A_data[i]<<endl;
//    }
//    cout<<endl;
//    cout<<"dimension:"<<dimension<<endl;
//    cout<<"B is "<<endl;
//    for(int i=0; i<dimension;i++)
//    {
//        cout<<matrix_B[i]<<endl;
//    }
//    cout<<"ptr_vec is "<<endl;
//    for(int i=0; i<int(ptr_vec.size());i++)
//    {
//        cout<<ptr_vec[i]<<endl;
//    }
    


    long long int *d_ptr_vector,*d_matrixA_col,*d_matrixA_data,*d_matrixB,*d_multiply_answer;

    long long int nonzero_size=matrix_A_nonzero_elem*sizeof(long long int);
    long long int dimension_size=(dimension*sizeof(long long int));

    hipMalloc((void **)&d_ptr_vector,((dimension+1)*sizeof(long long int)));
    hipMalloc((void **)&d_matrixA_col,nonzero_size);
    hipMalloc((void **)&d_matrixA_data,nonzero_size);
    hipMalloc((void **)&d_matrixB,dimension_size);
    hipMalloc((void **)&d_multiply_answer,dimension_size);


    hipMemcpy(d_ptr_vector,(&ptr_vec[0]),((dimension+1)*sizeof(long long int)),hipMemcpyHostToDevice);
    hipMemcpy(d_matrixA_col,(&matrix_A_col[0]),nonzero_size,hipMemcpyHostToDevice);
    hipMemcpy(d_matrixA_data,(&matrix_A_data[0]),nonzero_size,hipMemcpyHostToDevice);
    hipMemcpy(d_matrixB,(&matrix_B[0]),dimension_size,hipMemcpyHostToDevice);
    hipMemcpy(d_multiply_answer,(&matrix_C[0]),dimension_size,hipMemcpyHostToDevice);

    // long long int num_blocks;

    // num_blocks=(dimension/(long long int)(32));

    // if(num_blocks%32!=0)
    // {
    //     num_blocks+=1;
    // }

    spmv_csr_vector_kernel<<<dimension,32>>>(dimension,d_ptr_vector,d_matrixA_col,d_matrixA_data,d_matrixB,d_multiply_answer);
    

    hipMemcpy((&matrix_C[0]),d_multiply_answer,dimension_size,hipMemcpyDeviceToHost);
    outfile();
    return 0;
}

